#include "hip/hip_runtime.h"
#include <nbody/cuda_body.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

// #define DEBUG

template <typename... Args>
void UNUSED(Args &&...args [[maybe_unused]]) {}

__device__ __managed__ static float gravity = 100;
__device__ __managed__ static float space = 800;
__device__ __managed__ static float radius = 5;
__device__ __managed__ static int bodies = 200;
__device__ __managed__ static float elapse = 0.1;
__device__ __managed__ static float max_mass = 50;
__device__ __managed__ BodyPool *pool;

struct idx_struct
{
    size_t st_idx, end_idx;
};
__device__ __managed__ idx_struct_arr[2000];

__host__ void init_idx_struct_list(size_t thread_num)
{
    size_t idx_per_thread = pool->size / thread_num;
    size_t remainder = pool->size % thread_num;
    size_t st_idx = 0;
    for (size_t i = 0; i < threads.size(); i++)
    {
        size_t end_idx = i < remainder ? st_idx + idx_per_thread + 1 : st_idx + idx_per_thread;
        idx_struct_arr[i] = idx_struct{st_idx, end_idx};
        st_idx = end_idx;
    }
}
__global__ void worker()
{
    size_t i = threadIdx.x;
    size_t st_idx = idx_struct_arr[i].st_idx;
    size_t end_idx = idx_struct_arr[i].end_idx;
#ifdef DEBUG
    // printf("threadIdx: %d \n", i);
#endif
    for (size_t i = st_idx; i < end_idx; i++)
    {
        for (size_t j = 0; j < pool->size; ++j)
        {
            if (i == j)
                continue;
            pool->shared_memory_check_and_update(pool->get_body(i), pool->get_body(j), radius, gravity);
        }
    }
    __syncthreads();
    for (size_t i = st_idx; i < end_idx; i++)
    {
        pool->get_body(i).update_by_delta_var();
        pool->get_body(i).update_for_tick(elapse, space, radius);
    }
}

int main(int argc, char **argv)
{
    if (argc < 4)
    {
        std::cout << "Usage: cuda <size> <rounds> <thread_num>" << std::endl;
        return 0;
    }
    size_t rounds, thread_num;
    bodies = atoi(argv[1]);
    rounds = atoi(argv[2]);
    thread_num = atoi(argv[3]);
    init_idx_struct_list(thread_num);
    pool = new BodyPool(static_cast<size_t>(bodies), space, max_mass);
    dim3 grid(1);
    dim3 block(pool->size);
    using namespace std::chrono;
    auto begin = high_resolution_clock::now();
    for (size_t i = 0; i < rounds; i++)
    {
        pool->clear_acceleration();
        worker<<<grid, block>>>();
        hipDeviceSynchronize();
    }
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<nanoseconds>(end - begin).count() / rounds;
    std::cout << "block size: " << grid.x << std::endl;
    std::cout << "threads per block: " << block.x << std::endl;
    std::cout << "problem size: " << pool->size << std::endl;
    std::cout << "duration(ns/round): " << duration << std::endl;
    std::cout << "rounds: " << rounds << std::endl;
#ifdef DEBUG
    printf("pool size: %zd\n", pool->size);
    for (auto &each : pool->x)
        std::cout << each << " ";
    std::cout << std::endl;
    for (auto &each : pool->vx)
        std::cout << each << " ";
    std::cout << std::endl;
    for (auto &each : pool->ax)
        std::cout << each << " ";
    std::cout << std::endl;
#endif
    delete pool;
    hipDeviceReset();
    return 0;
}
